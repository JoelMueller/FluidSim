#include "hip/hip_runtime.h"
#include "stdint.h"
#include "Array2D.cu"

extern __device__ float clamp(float x, float minX, float maxX){
	return max(minX, min(maxX, x));
}
extern __device__ int clamp(int x, int minX, int maxX){
	return max(minX, min(maxX, x));
}

extern "C" __global__ void advect(Array2D<0> q, Array2D<1> qNew, Array2D<2> u, Array2D<3> v, const float dt, const float rdx)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	int height = qNew.getCount(0);
	int width = qNew.getCount(1);
	
	if (i >= width || j >= height || i < 0 || j < 0)
		return;

	float pos_x = i - u[j][i] * dt * rdx;
	float pos_y = j - v[j][i] * dt * rdx;
	pos_x = clamp(pos_x, 0.0, (float)width-1);
	pos_y = clamp(pos_y, 0.0, (float)height-1);
	int x = (int) floor(pos_x);
	int y = (int) floor(pos_y);
	float t_x = pos_x - x;
	float t_y = pos_y - y;


	// bilinear interpolation
	float pixel00 = q[y][x];
	float pixel10 = q[y][clamp((x + 1), 0, width - 1)];
	float pixel01 = q[clamp((y + 1), 0, height - 1)][x];
	float pixel11 = q[clamp((y + 1), 0, height - 1)][clamp((x + 1), 0, width - 1)];

	qNew[j][i] = (1.f - t_y)*((1.f - t_x)*pixel00 + t_x*pixel10) + t_y*((1.f - t_x)*pixel01 + t_x*pixel11);
}

extern "C" __global__ void jacobi(Array2D<0> x, Array2D<1> xNew, Array2D<2> b, const float alpha, const float rbeta)
{
	size_t i = blockIdx.x * blockDim.x + threadIdx.x;
	size_t j = blockIdx.y * blockDim.y + threadIdx.y;
	int height = xNew.getCount(0);
	int width = xNew.getCount(1);

	if (i >= width || j >= height || i < 0 || j  < 0)
		return;

	xNew[j][i] = rbeta * (alpha * b[j][i]
		+ x[j][clamp((i + 1), 0, width - 1)]
		+ x[j][clamp((i - 1), 0, width - 1)]
		+ x[clamp((j + 1), 0, height - 1)][i]
		+ x[clamp((j - 1), 0, height - 1)][i]);
}

extern "C" __global__ void divergence(Array2D<0> u, Array2D<1> v, Array2D<2> div, const float halfrdx)
{
	size_t i = blockIdx.x * blockDim.x + threadIdx.x;
	size_t j = blockIdx.y * blockDim.y + threadIdx.y;
	int height = div.getCount(0);
	int width = div.getCount(1);

	if (i >= width || j >= height || i < 0 || j < 0)
		return;

	div[j][i] = halfrdx * (u[j][clamp(i + 1, 0, width - 1)]
		- u[j][clamp(i - 1, 0, width - 1)]
		+ v[clamp(j + 1, 0, height - 1)][i]
		- v[clamp(j - 1, 0, height - 1)][i]);
}

extern "C" __global__ void subtractGradient(Array2D<0> p, Array2D<1> u, Array2D<2> v, Array2D<3> uNew, Array2D<4> vNew, const float halfrdx)
{
	size_t i = blockIdx.x * blockDim.x + threadIdx.x;
	size_t j = blockIdx.y * blockDim.y + threadIdx.y;
	int height = uNew.getCount(0);
	int width = uNew.getCount(1);

	if (i >= width || j >= height || i < 0 || j < 0)
		return;

	uNew[j][i] = u[j][i] - halfrdx * (p[j][clamp(i + 1, 0, width - 1)]
		- p[j][clamp(i - 1, 0, width - 1)]);
	vNew[j][i] = v[j][i] - halfrdx * (p[clamp(j + 1, 0, height - 1)][i]
		- p[clamp(j - 1, 0, height - 1)][i]);
}


extern "C" __global__ void boundary(Array2D<0> x,  float scale)
{
	size_t i = blockIdx.x * blockDim.x + threadIdx.x;
	size_t j = blockIdx.y * blockDim.y + threadIdx.y;
	int height = x.getCount(0);
	int width = x.getCount(1);

	if (i >= width || j >= height)
		return;

	if (i == 0)
		x[j][i] = scale*x[j][i + 1];
	else if (i == width - 1)
		x[j][i] = scale*x[j][i - 1];
	else if (j == 0)
		x[j][i] = scale*x[j + 1][i];
	else if (j == height - 1)
		x[j][i] = scale*x[j - 1][i];
}

extern "C" __global__ void addInk(Array2D<0> u, Array2D<1> v, Array2D<2> ink, const int x, const int y, const float u_, const float v_, const float ink_)
{
	size_t i = blockIdx.x * blockDim.x + threadIdx.x;
	size_t j = blockIdx.y * blockDim.y + threadIdx.y;
	int height = u.getCount(0);
	int width = u.getCount(1);

	if (i >= width || j >= height || i < 0 || j < 0)
		return;
	
	int dx = i - x;
	int dy = j - y;
	float s = 1.f / pow(2., static_cast<double>(dx*dx + dy*dy) / 200.);

	u[j][i] += u_ * s;
	v[j][i] += v_ * s;
	ink[j][i] += ink_ * s;
	ink[j][i] = clamp(ink[j][i], 0.0, 255.0);
}

extern "C" __global__ void convertToColor(uint8_t *color, Array2D<0> x)
{
	size_t i = blockIdx.x * blockDim.x + threadIdx.x;
	size_t j = blockIdx.y * blockDim.y + threadIdx.y;
	int height = x.getCount(0);
	int width = x.getCount(1);
	size_t index = i + width * j;

	if (i >= width || j >= height || i < 0 || j < 0)
		return;

	uint8_t value = 255 - static_cast<uint8_t>(x[j][i]);
	color[4 * index] = value;
	color[4 * index + 1] = value;
	color[4 * index + 2] = value;
	color[4 * index + 3] = 0;
}

extern "C" __global__ void convertToColor2(uint8_t *color, Array2D<0> r, Array2D<1> g, Array2D<2> b)
{
	size_t i = blockIdx.x * blockDim.x + threadIdx.x;
	size_t j = blockIdx.y * blockDim.y + threadIdx.y;
	int height = r.getCount(0);
	int width = r.getCount(1);
	size_t index = i + width * j;

	if (i >= width || j >= height || i < 0 || j < 0)
		return;

	color[4 * index] = static_cast<uint8_t>(r[j][i]);
	color[4 * index + 1] = static_cast<uint8_t>(g[j][i]);
	color[4 * index + 2] = static_cast<uint8_t>(b[j][i]);
	color[4 * index + 3] = 0;
}